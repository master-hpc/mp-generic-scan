#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>


__global__ void prefixSumKernel(int *v, int n)
{
	int step = 1;
	while (step < n) {

		int indice1 = threadIdx.x;
		int indice2 = threadIdx.x + step;

		if (indice2 < n) {
			v[indice2] = v[indice1] + v[indice2];
		}

		step = step * 2;
	}

}

int main()
{
	const int arraySize = 10;
	int v[arraySize] = { 3, 7, 1, 10, 6, 9, 5, 2, 8, 4 };

	int *dev_v = 0;

	hipMalloc((void**)&dev_v, arraySize * sizeof(int));

	hipMemcpy(dev_v, v, arraySize * sizeof(int), hipMemcpyHostToDevice);

	prefixSumKernel << <1, arraySize - 1 >> >(dev_v, arraySize);

	hipMemcpy(v, dev_v, arraySize * sizeof(int), hipMemcpyDeviceToHost);

	hipFree(dev_v);

	for (int i = 0; i < arraySize; i++) {
		printf(" %d ", v[i]);
	}

	return 0;
}

