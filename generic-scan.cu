
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void scan(int* v, const int n);

int main(int argc, char** argv) {

	const int size = 10;

	int h_v[size] = { 3, 7, 1, 10, 6, 9, 5, 2, 8, 4 };

	int *d_v = 0;

	hipMalloc((void**)&d_v, size * sizeof(int));

	hipMemcpy(d_v, h_v, size * sizeof(int), hipMemcpyHostToDevice);

	dim3 grdDim(1, 1, 1);
	dim3 blkDim(size - 1, 1, 1);	

	scan <<<grdDim, blkDim>>>(d_v, size);

	hipMemcpy(h_v, d_v, size * sizeof(int), hipMemcpyDeviceToHost);

	hipFree(d_v);

	for (int i = 0; i < size; i++) {
		std::cout << (i == 0 ? "{" : "") <<  h_v[i] << (i < size -1 ? " ," : "}");
	}
	std::cout << std::endl;

	return 0;
}


__global__ void scan(int *v, const int n)
{
        int tIdx = threadIdx.x;
	int step = 1;
	
        while (step < n) {

                int indiceDroite = tIdx;
                int indiceGauche = indiceDroite + step;

                if (indiceGauche < n) {
                        v[indiceDroite] = v[indiceDroite] + v[indiceGauche];
                }

                step = step * 2;
		__syncthreads();

        }

}
